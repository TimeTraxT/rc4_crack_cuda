#include "hip/hip_runtime.h"
#include "rc4.h"
/************************************************************************/
/* 
������˼·��ÿ�λ�ȡһ����Կ�����ܶ�Ӧ�����ģ����õ��������Ƿ�����ĳ��������
���ǹ�������Ҫ���м����̫���ˣ�ת��һ�룬���ĺ����������Ĺ�ϵ����ô��֪��
�ĺ��������Ļ����ܵõ���Կ����ĳЩλ�õ�ֵ�������Ϳ���ʡȥ���ٿռ�~~
*/
/************************************************************************/
/** 
 * \brief,to generate the candidate key 
 **/
__device__ bool generateKey(int startIndex)
{
	unsigned char currentkeyLen=shared_mem[startIndex+KEY_LEN_OFFSET],tempP=currentkeyLen;
	if(currentkeyLen<MAX_KEY_LENGTH)
	{
		shared_mem[startIndex+currentkeyLen]++;
		while(shared_mem[startIndex+tempP]>END_CHARACTER&&tempP>0)
		{
			shared_mem[startIndex+tempP]=START_CHARACTER;
			tempP--;
			shared_mem[startIndex+tempP]++;
		}
		if(shared_mem[startIndex]>END_CHARACTER)
		{
			currentkeyLen++;
			shared_mem[startIndex]=START_CHARACTER;
			shared_mem[startIndex+currentkeyLen]=START_CHARACTER;
		}
		shared_mem[startIndex+KEY_LEN_OFFSET]=currentkeyLen;
		shared_mem[startIndex+currentkeyLen+1]='\0';
		return true;
	}
	return false;
}

__device__ unsigned char* genKey(unsigned char*res,unsigned long long val,int*key_len)
{
	char p=MAX_KEY_LENGTH-1;
	while (val&&p>=0) {
		res[p--] = (val - 1) % KEY + START_CHARACTER;
		val = (val - 1) / KEY;
	}
	*key_len=(MAX_KEY_LENGTH-p-1);
	return res+p+1;
}

__global__ void crackRc4Kernel(unsigned char*key, volatile bool *found)
{
	if(*found) asm("exit;");

	int bdx=blockIdx.x, tid=threadIdx.x, keyLen=0;

	const unsigned long long cycleNum=maxNum/(gridDim.x*blockDim.x*OPERATE_KEY_PER_THREAD);

	unsigned long long startPoint;
	bool justIt=true;

	unsigned char tempArray[MAX_KEY_LENGTH+1];
	unsigned char * vKey;

	for (unsigned long long i=0;i<=cycleNum&startPoint<maxNum;i++)
	{
		if(*found) asm("exit;");

		startPoint=i*(gridDim.x*blockDim.x*OPERATE_KEY_PER_THREAD);
		if(startPoint==0) startPoint=1;
		vKey=genKey(tempArray,startPoint,&keyLen);
		memcpy((shared_mem+MEMEORY_PER_THREAD*tid),vKey,keyLen);
		keyLen--;
		shared_mem[MEMEORY_PER_THREAD*tid+KEY_LEN_OFFSET]=keyLen;
		for (int j=0;j<OPERATE_KEY_PER_THREAD;j++)
		{
			if(*found) asm("exit;");
			if(j!=0) generateKey(MEMEORY_PER_THREAD*tid);

			keyLen=shared_mem[MEMEORY_PER_THREAD*tid+KEY_LEN_OFFSET];
			vKey=shared_mem+MEMEORY_PER_THREAD*tid;

			if(*found) asm("exit;");

			justIt=device_isKeyRight(vKey,keyLen+1,found);

			//��ǰ��Կ��������
			if (!justIt) continue;

			//�ҵ��Ļ��˳�
			if(*found) asm("exit;");

			//�ҵ�ƥ����Կ��д��Host����������,�޸�found,�˳�����
			*found=true;
			memcpy(key,vKey,keyLen+1);
			key[keyLen+1]=0;
			__threadfence();
			asm("exit;");
			break;
		}		
	}
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t crackRc4WithCuda(unsigned char* knownKeyStream_host, int knownStreamLen_host, unsigned char*key, bool*found)
{
	unsigned char *key_dev ;
	bool* found_dev;
	hipError_t cudaStatus;


	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	cudaStatus = hipMalloc((void**)&key_dev, (MAX_KEY_LENGTH+1) * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&found_dev, sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	//�����Ƿ��ҵ���Կ����
	cudaStatus = hipMemcpy(found_dev, found, sizeof(bool), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	//���Ƴ����ڴ�
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(knowStream_device), knownKeyStream_host,sizeof(unsigned char)*knownStreamLen_host);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpyToSymbol failed!");
		goto Error;
	}

	cudaStatus = hipMemcpyToSymbol((const void *)&knownStreamLen_device,(const void *)&knownStreamLen_host,sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpyToSymbol failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	int threadNum=floor((double)(prop.sharedMemPerBlock/MEMEORY_PER_THREAD)),share_memory=prop.sharedMemPerBlock;
	if(threadNum>MAX_THREAD_NUM){
		threadNum=MAX_THREAD_NUM;
		share_memory=threadNum*MEMEORY_PER_THREAD;
	}
	crackRc4Kernel<<<BLOCK_NUM, threadNum, share_memory>>>(key_dev,found_dev);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(key, key_dev, (MAX_KEY_LENGTH+1) * sizeof(unsigned char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(found, found_dev,  sizeof(bool), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(key_dev);
	hipFree(found_dev);

	return cudaStatus;
}

int main(int argc, char *argv[])
{
//	printf("%c",0x7d);
	unsigned char* s_box = (unsigned char*)malloc(sizeof(unsigned char)*256);
	//��Կ
	unsigned char encryptKey[]="!}";
	//����
	unsigned char buffer[] = "Life is a chain of moments of enjoyment, not only about survivalO(��_��)O~";
	int buffer_len=strlen((char*)buffer);
	prepare_key(encryptKey,strlen((char*)encryptKey),s_box);
	rc4(buffer,buffer_len,s_box);	

	unsigned char knownPlainText[]="Life";
	int known_p_len=strlen((char*)knownPlainText);
	unsigned char* knownKeyStream=(unsigned char*)malloc(sizeof(unsigned char)*known_p_len);
	for (int i=0;i<known_p_len;i++)
	{
		knownKeyStream[i]=knownPlainText[i]^buffer[i];
	}

	unsigned char * key=(unsigned char*)malloc( sizeof(unsigned char) * (MAX_KEY_LENGTH+1));

	hipEvent_t start,stop;
	hipError_t cudaStatus=hipEventCreate(&start);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventCreate(start) failed!");
		return 1;
	}
	cudaStatus=hipEventCreate(&stop);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventCreate(stop) failed!");
		return 1;
	}

	cudaStatus=hipEventRecord(start,0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventRecord(start) failed!");
		return 1;
	}

	bool found=false;
	cudaStatus = crackRc4WithCuda(knownKeyStream, known_p_len , key, &found);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	cudaStatus=hipEventRecord(stop,0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventRecord(stop) failed!");
		return 1;
	}

	cudaStatus=hipEventSynchronize(stop);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventSynchronize failed!");
		return 1;
	}
	float useTime;
	cudaStatus=hipEventElapsedTime(&useTime,start,stop);
	useTime/=1000;
	printf("The time we used was:%fs\n",useTime);
	if (found)
	{
		printf("The right key has been found.The right key is:%s\n",key);
		prepare_key(key,strlen((char*)key),s_box);
		rc4(buffer,buffer_len,s_box);
		printf ("\nThe clear text is:\n%s\n",buffer);
	}

	hipEventDestroy(start);
	hipEventDestroy(stop);

	free(key);
	free(knownKeyStream);
	free(s_box);
	hipDeviceReset();
	return 0;
}



